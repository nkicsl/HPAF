#include "hip/hip_runtime.h"
#include <mat.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <matrix.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "cuda_util.h"
using namespace std;

const int FILTER_NUM = 20;//number of filters
const int P_NUM = 19;//number of layers of each convolution operation
const int LEAP = 2;//leap size
const int GP_NUM = 2;//size of each group
const int NEU_NUM1 = 100;//number of neurons in full connection layer
const int NEU_NUM2 = 13;
const int NEIGHBOR = 8;//number of neighbor pixels
double LEARN_RATE = 0.008;
const double MIN_ERR = 0.0001;
const int VALID_BATCH = 5;
const int DATA_BATCH =10;

//Initialize CUDA
bool InitCUDA(){
	int count;
	hipGetDeviceCount(&count);
	if(count==0){
		fprintf(stderr,"There is no device.\n");
		return false;
	}
	int i;
	for (i =0; i<count;i++){
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop,i)==hipSuccess){
			if(prop.major>=1){       
                break;
			}
		}
	}
	if(i==count){
		fprintf(stderr,"There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(4);
	return true;
}


//copy data to shared memory
__device__ void copy_data_to_shared(double * data, double * data_tmp, int tid, int offset, int head, int length)
{
	for(size_t i = tid * offset; i < (tid + 1) * offset && (i < length); i++)
    {
		data_tmp[i] = data[i + head];
	}
	__syncthreads();

}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// forward propagation kernels
// forward convolution
__global__ static void convolution(int data_id,int batch_id, double * train,double * filters, double * re,double * bias, int z,int re_size)
{

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	if(tid<re_size && bid < FILTER_NUM)
	{
		extern __shared__ double train_tmp[];
        int head = data_id * (NEIGHBOR + 1) * z;
		int length = (NEIGHBOR+1) * z;
        int offset = (length - 1)/re_size + 1;
		copy_data_to_shared(train,train_tmp, tid, offset, head, (NEIGHBOR+1) * z);
        
		__shared__ double filters_tmp[(NEIGHBOR+1) * P_NUM];
		head = bid * (NEIGHBOR+1)*P_NUM;
		length = (NEIGHBOR+1)*P_NUM;
		offset = (length - 1)/re_size + 1;
		copy_data_to_shared(filters, filters_tmp,tid,offset,head,length);
		__syncthreads();

		double mid = 0;
		for(size_t i = 0; i < (NEIGHBOR + 1) * P_NUM; i++){
            mid = mid + filters_tmp[i] * train_tmp[tid * (NEIGHBOR+1) * LEAP + i];
		}
		mid = mid + bias[bid];
		re[tid + bid * re_size + batch_id * re_size * FILTER_NUM] = 2/(1 + (1/exp(2*mid))) - 1;
	}
}

// forward maxpooling
__global__ static void maxpooling(int batch_id,double * re,double * mre,int * mre_index,int re_size,int mre_num)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid < mre_num && bid < FILTER_NUM){
		double mid;
		int mid_index, head, tail;
		head = tid * GP_NUM + bid * re_size + batch_id * re_size * FILTER_NUM;
		tail = (tid+1) * GP_NUM + bid * re_size + batch_id * re_size * FILTER_NUM;
		mid = re[head];
		mid_index = head;
		for(int i=head; i<tail && (i<(bid+1)*re_size+batch_id*re_size*FILTER_NUM); i++){
			if(mid < re[i]){
				mid = re[i];
				mid_index=i;
			}
		}
		mre[tid + bid*mre_num + batch_id*mre_num*FILTER_NUM] = mid;
		mre_index[tid + bid*mre_num + batch_id*mre_num*FILTER_NUM] = mid_index;
	}
}

// forward fully connection
__global__ static void fully_connect(int batch_id,double * mre,double * omega,double * bias,double * F1,int mre_size){

	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid<mre_size && bid<NEU_NUM1){
		extern __shared__ double ner[];
		ner[tid] = omega[bid + tid*NEU_NUM1] * mre[tid + batch_id*mre_size];
		__syncthreads();//waiting for other threads

		int length = mre_size;
		int offset = (length - 1)/2 + 1;

		while(length >= 2)
		{
			if(tid + offset < length)
			{
				ner[tid] = ner[tid] + ner[tid + offset];
			}
			offset = (offset - 1)/2 + 1;
			length = (length - 1)/2 + 1;
			__syncthreads();
		}

		F1[bid + batch_id*NEU_NUM1] = 2/(1 + 1/exp((ner[0] + bias[bid]) * 2)) - 1;
	}
}

// forward output
__global__ static void output(int batch_id, double * F1, double * omega2, double * bias, double * O2){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < NEU_NUM2){
		//copy F1 to shared memory
		__shared__ double F1_tmp[NEU_NUM1];
		int offset = (NEU_NUM1-1)/NEU_NUM2 + 1;
		copy_data_to_shared(F1, F1_tmp, id, offset, batch_id*NEU_NUM1, NEU_NUM1);
		__syncthreads();
		__shared__ double O2_tmp[1][NEU_NUM2];

		//compute the output of a neuron
		double mid = 0;
		for(int i=0; i<NEU_NUM1; i++){
			mid = mid + omega2[id + i*NEU_NUM2] * F1_tmp[i];
		}
		O2[id + batch_id*NEU_NUM2] = exp(mid+ bias[id]);
		O2_tmp[0][id] = O2[id + batch_id*NEU_NUM2];
		__syncthreads(); //waiting for other threads

		int length = NEU_NUM2;//length of the array needed to be summed up
		offset = (length - 1)/2 +1;//bias value
		while(length >= 2)
		{
			if(id + offset < length){
				O2_tmp[0][id] = O2_tmp[0][id] + O2_tmp[0][id + offset];
			}
			offset = (offset - 1)/2 + 1;
			length = (length - 1)/2 + 1;
			__syncthreads();//waiting for all threads
		}
		O2[id + batch_id*NEU_NUM2] = O2[id + batch_id*NEU_NUM2]/O2_tmp[0][0];

	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// backward propagation kernels
// output layer
__global__ static void bp_output(int iter, int train_idx, int batch_id, double LEARN_RATE, double * labels, double * O2, double * delta_L_z)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;
	if(id < NEU_NUM2){
		delta_L_z[id + batch_id*NEU_NUM2] = (O2[id + batch_id*NEU_NUM2] - labels[id + train_idx * NEU_NUM2])/NEU_NUM2;
	}
}

// fully_connect
__global__ static void bp_fully_connect(int iter, int batch_id, double LEARN_RATE, double * omega2, double * F1, double * delta_L_z, double *delta_f_w, double * delta_f_z)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid<NEU_NUM2 && bid<NEU_NUM1){
		__shared__ double dfa[1][NEU_NUM2];
		dfa[0][tid] = omega2[tid + bid*NEU_NUM2] * delta_L_z[tid + batch_id*NEU_NUM2];
		__syncthreads();

		delta_f_w[tid + bid*NEU_NUM2 + batch_id*NEU_NUM1*NEU_NUM2] = F1[bid + batch_id*NEU_NUM1] * delta_L_z[tid + batch_id*NEU_NUM2]; 

		int length = NEU_NUM2;
		int offset = (length - 1)/2 + 1;
		while(length >= 2){
			if(tid + offset < length){
				dfa[0][tid] = dfa[0][tid] + dfa[0][tid+offset];
			}
			length = (length - 1)/2 + 1;
			offset = (offset - 1)/2 + 1;
			__syncthreads();
		}

		delta_f_z[bid + batch_id*NEU_NUM1] = dfa[0][0] * (1 + F1[bid +batch_id*NEU_NUM1]) * (1 - F1[bid + batch_id*NEU_NUM1]);
	}
}

// maxpooling layer
__global__ static void bp_maxpooling(int iter, int mre_size, int re_size, int batch_id, double LEARN_RATE, int * mre_index, double * omega1, double * mre, double * delta_f_z, double * delta_m_w,  double * delta_22)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid<NEU_NUM1 && bid<mre_size){
		__shared__ double mid[1][NEU_NUM1];
		mid[0][tid] = omega1[tid + bid*NEU_NUM1] * delta_f_z[tid + batch_id*NEU_NUM1];
		__syncthreads();//waiting for all threads
		int length = NEU_NUM1;
		int offset = (length - 1)/2 + 1;
		while(length >= 2){
			if(tid + offset < length){
				mid[0][tid] = mid[0][tid] + mid[0][tid+offset];
			}
			length = (length - 1)/2 + 1;
			offset = (offset - 1)/2 + 1;
			__syncthreads();
		}

		delta_m_w[tid + bid*NEU_NUM1 + batch_id*mre_size*NEU_NUM1] = mre[bid + batch_id*mre_size] * delta_f_z[tid + batch_id*NEU_NUM1];

		if(tid < 1)
		delta_22[mre_index[bid + batch_id*mre_size] ] =mid[0][0]* (1 + mre[bid + batch_id*mre_size]) * (1 - mre[bid + batch_id*mre_size]);
	}
}

// convolutional layer
__global__ static void bp_convolution(int iter, int i0, int batch_id, double LEARN_RATE, int z, int mre_num,int re_size, int * mre_index, double * delta_22, double * delta_k_w, double * delta_k_b, double * data)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid < ((NEIGHBOR+1)*P_NUM) && (bid < FILTER_NUM))
	{
		int head = i0*(NEIGHBOR+1)*z;
		int length = (NEIGHBOR+1)*z;
		int offset = (length - 1)/((NEIGHBOR+1)*P_NUM) + 1;
		extern __shared__ double train_tmp[];
		copy_data_to_shared(data, train_tmp, tid, offset, head, length);
        __syncthreads();

        __shared__ double delta_kw[1][(NEIGHBOR+1)*P_NUM];
		delta_kw[0][tid] = 0;

		int idx, n, i, h;
		double mid = 0;
		for(i=0; i<mre_num; i++){
			idx = mre_index[i + bid*mre_num + batch_id*mre_num*FILTER_NUM];
			n = idx % re_size;
			h = n*(NEIGHBOR+1)*LEAP;
			delta_kw[0][tid] = delta_kw[0][tid] + delta_22[idx ] * train_tmp[tid + h];
			mid = mid + delta_22[idx ];
		}

		delta_k_w[tid + bid*(NEIGHBOR+1)*P_NUM + batch_id*(NEIGHBOR+1)*P_NUM*FILTER_NUM] =delta_kw[0][tid]/mre_num;
		
		if(tid < 1)
			delta_k_b[bid + batch_id*FILTER_NUM] = (mid/mre_num);
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// update params kernels
// output layer
__global__ static void update_output(int iter,int batch_size, double LEARN_RATE, double * delta_L_z, double * bias2)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid < batch_size && bid<NEU_NUM2){
		__shared__ double mid[1][DATA_BATCH];
		mid[0][tid] = delta_L_z[bid + tid*NEU_NUM2];
		__syncthreads();//waiting for all threads

		int length = batch_size;
		int offset = (length - 1)/2 + 1;
		while(length >= 2){
			if(tid + offset < length){
				mid[0][tid] = mid[0][tid] + mid[0][tid + offset];
			}
			length = (length - 1)/2 + 1;
			offset = (offset - 1)/2 + 1;
			__syncthreads();
		}

		if(tid < 1)
			bias2[bid] = bias2[bid] - mid[0][0]*LEARN_RATE/batch_size;
	}
}
// fully_connect layer
__global__ static void update_fully_connect(int batch_size, double LEARN_RATE, double * omega2,double * bias1,double *delta_f_w, double * delta_f_z)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid<NEU_NUM2 && bid<NEU_NUM1){
		double mid0 = 0, mid1 = 0;
		for(int i=0; i<batch_size; i++){
			mid0 = mid0 + delta_f_w[tid + bid*NEU_NUM2 + i*NEU_NUM1*NEU_NUM2];
			mid1 = mid1 + delta_f_z[bid + i*NEU_NUM1];
		}
		omega2[tid + bid*NEU_NUM2] = omega2[tid + bid*NEU_NUM2] - LEARN_RATE * mid0/batch_size;
		
		if(tid < 1){
			bias1[bid] = bias1[bid] - LEARN_RATE * mid1 / batch_size;
		}
	}
}
// maxpooling layer
__global__ static void update_maxpooling(int iter, int mre_size, int batch_size, double LEARN_RATE, double * omega1, double * delta_m_w)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid<NEU_NUM1 && bid<mre_size){
		double mid=0;
		for(int i=0; i<batch_size; i++){
			mid = mid + delta_m_w[tid + bid*NEU_NUM1 + i*mre_size*NEU_NUM1];
		}
		
		omega1[tid + bid*NEU_NUM1] = omega1[tid + bid*NEU_NUM1] - LEARN_RATE*mid/batch_size;
	}
}

// convolution layer
__global__ static void update_convolution(int batch_size, int re_size, double LEARN_RATE, double * delta_22, double * delta_k_w, double * delta_k_b, double * filters, double * bias0)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	if(tid < ((NEIGHBOR+1)*P_NUM) && (bid < FILTER_NUM))
	{
		double mid0 = 0, mid1 = 0;
		for(int i=0; i<batch_size; i++){
			mid0 = mid0 + delta_k_w[tid + bid*(NEIGHBOR+1)*P_NUM +i*(NEIGHBOR+1)*P_NUM*FILTER_NUM];
			mid1 = mid1 + delta_k_b[bid + i*FILTER_NUM];
		}
		filters[tid + bid*(NEIGHBOR+1)*P_NUM] = filters[tid + bid*(NEIGHBOR+1)*P_NUM] - LEARN_RATE*mid0/batch_size;
		
		if(tid < 1){
			bias0[bid] = bias0[bid] - LEARN_RATE*mid1/batch_size;
		}	
	}
}

__global__ static void loss_function(int batch_id, int batch_size, double * O2, double * labels, double * loss_values)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    double tmp = 0.0;
    if(tid < batch_size){
        for(size_t i=0; i<NEU_NUM2; i++){
            tmp = tmp + labels[i + (batch_id * DATA_BATCH + tid) * NEU_NUM2] * log(O2[i + tid * NEU_NUM2]) +
                  (1 - labels[i + (batch_id * DATA_BATCH + tid) * NEU_NUM2]) * log(1 - O2[i + tid * NEU_NUM2]);
        }

        loss_values[tid] = -tmp/NEU_NUM2;
    }
}

//preprocessing
__global__ static void processing(int iter, double * data, int * train_index, double * processed_data, int x, int y, int z, int train_size)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int threadNum = blockDim.x * gridDim.x;
	int id = tid + iter * threadNum;

	//int idx = id * (NEIGHBOR+1) * z;
	if (id < train_size){
		int idx = id * (NEIGHBOR+1) * z;
		int i, j;
		for (i=0; i<z; i++){
			for (j=0; j<(NEIGHBOR+1); j++){
				processed_data[idx] = data[train_index[j + id*(NEIGHBOR+1)] + i * x*y];
				idx = idx + 1;	
			}
		}
	}
}

double lossfunction(double * output, double * labels, int idx){
	double l = 0;
	int i;
	for(i=0; i<NEU_NUM2; i++){
		l = l + labels[i + idx*NEU_NUM2] * log(output[i]) + (1 - labels[i + idx*NEU_NUM2])*log(1 - output[i]); 
        //(output[i] - labels[i + idx*NEU_NUM2]) * (output[i] - labels[i + idx*NEU_NUM2]);
	}
	l = -l/NEU_NUM2;
	return l;
}


//compute correct rate
double count_err(double * test_labels, double * output, int test_idx)
{
	double right=0;
	double max =0;
	int idx = 0;
	int i;
	for(i=0; i<NEU_NUM2; i++){
		if(output[i]>max){
			max = output[i];
			idx = i;
		}
	}
	if((idx+1) == int(test_labels[test_idx]))
		right = 1;
	
	return right;
}

//Insert current loss value to the queue
void insert_line(double * a, double b){
	for(int i=1; i<VALID_BATCH; i++){
		a[i-1] = a[i];
	}
	a[VALID_BATCH-1] = b;
}

//shuffle
void shuffle(int * data, double * labels, int dim_row, int width){
	int index,  i;
	int temp;
	double tmp;
	srand(time(NULL));
	for(i=0; i<width; i++){
		index=rand()%(width-i) + i;
		if(index != i){
			for(int j=0; j<dim_row; j++){
				temp = data[j + i*dim_row];
				data[j + i*dim_row] = data[j +index*dim_row];
				data[j + index*dim_row] = temp;
			}

			for(int j=0; j<NEU_NUM2; j++){
				tmp = labels[j + i*NEU_NUM2];
				labels[j + i*NEU_NUM2] = labels[j + index*NEU_NUM2];
				labels[j + index*NEU_NUM2] = tmp;
			}
		}
	}
}

//training
double training(double * data, double * labels, int x, int y, int z){
	clock_t start, end;
	start = clock();	
	double * gpu_data;//original hyperspectral image, saved in global memory
	double * gpu_processed_train;//extracted train samples
	double * gpu_processed_test;//extracted test samples
	int * gpu_train_index;//index of train samples and their neighbors
	int * gpu_test_index;//index of test samples
	double * gpu_processed_labels;//encoded train labels

	//preprocessing
	int data_size = 0;
	int * data_index = new int [x*y];
	for(int i=0; i<x*y; i++){
		if(labels[i] != 0){
			data_index[data_size]=i;
			data_size ++;
		}
	}
	int test_size = (data_size-1)/5 + 1;
	int train_size = data_size - test_size;
	fprintf(stdout,"train_size:%d  test_size:%d\n",train_size,test_size);
	int * train_index = new int [train_size * (NEIGHBOR + 1)];
	int * test_index = new int [test_size * (NEIGHBOR+1)];

	double * processed_labels = new double [train_size * NEU_NUM2]();
	double * test_labels = new double [test_size]();

	int tr=0, te=0;
		for (int i=0; i<data_size; i++){
		if (i%5 != 0){
			train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1)] = data_index[i];//index of current labeled pixel
			if(NEIGHBOR == 4)
			{
				train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) - 1] = data_index[i] - 1;
				train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) + 1] = data_index[i] + 1;
				train_index[0 + tr * (NEIGHBOR+1)] = data_index[i] - x;
				train_index[NEIGHBOR + tr * (NEIGHBOR+1)] = data_index[i] + x;
				

				if((data_index[i] % x) == 0){//first row
					train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) - 1] = train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) + 1];
				}
				if((data_index[i] % x) == (x-1)){//last row
					train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) + 1] = train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) - 1];
				}
				if((data_index[i]/x) == 0){//first column
					train_index[0 + tr * (NEIGHBOR+1)] = train_index[NEIGHBOR + tr * (NEIGHBOR+1)];
				}
				if((data_index[i]/x) == (y-1)){//last column
					train_index[NEIGHBOR + tr * (NEIGHBOR+1)] = train_index[0 + tr * (NEIGHBOR+1)];
				}
			}
			if(NEIGHBOR == 8)
			{
				train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) - 1] = data_index[i] - 1;
				train_index[(NEIGHBOR/2) + tr * (NEIGHBOR+1) + 1] = data_index[i] + 1;
				for(int j0=0;j0<3;j0++){
					train_index[j0 + tr * (NEIGHBOR+1)] = data_index[i] - 1 - x + j0;
					train_index[j0+6 + tr * (NEIGHBOR+1)] = data_index[i] - 1 + x + j0;
				}

				if((data_index[i] % x) == 0){//first row
					for (int j=0; j<3; j++)
						train_index[j*3 + tr*(NEIGHBOR+1)] = train_index[j*3+2 + tr*(NEIGHBOR+1)];
				}
				if((data_index[i] % x) == (x-1)){//last row
					for(int j=0;j<3;j++)
							train_index[j*3+2 + tr*(NEIGHBOR+1)] = train_index[j*3 + tr*(NEIGHBOR+1)];
				}
				if((data_index[i]/x) == 0){//first column
					for(int j=0;j<3;j++)
						train_index[j + tr*(NEIGHBOR+1)] = train_index[j+6 + tr*(NEIGHBOR+1)];
				}
				if((data_index[i]/x) == (y-1)){//last column
					for(int j=0;j<3;j++)
						train_index[j+6  + tr*(NEIGHBOR+1)] = train_index[j + tr*(NEIGHBOR+1)];
				}
			}

			int mid = int(labels[data_index[i]])-1 + tr*NEU_NUM2;
			processed_labels[mid] = 1;
			tr = tr + 1;
		}
		if(i%5 == 0){
			test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1)] = data_index[i];//index of current labeled pixel
			if(NEIGHBOR == 4)
			{
				test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) - 1] = data_index[i] - 1;
				test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) + 1] = data_index[i] + 1;
				test_index[0 + te * (NEIGHBOR+1)] = data_index[i] - x;
				test_index[NEIGHBOR+ te * (NEIGHBOR+1)] = data_index[i] + x;

				if((data_index[i] % x) == 0){//first row
					test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) - 1] = test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) + 1];
				}
				if((data_index[i] % x) == (x-1)){//last row
					test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) + 1] = test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) - 1];
				}
				if((data_index[i]/x) == 0){//first column
					test_index[0 + te * (NEIGHBOR+1)] = test_index[NEIGHBOR+ te * (NEIGHBOR+1)];
				}
				if((data_index[i]/x) == (y-1)){//last column
					test_index[NEIGHBOR+ te * (NEIGHBOR+1)] = test_index[0 + te * (NEIGHBOR+1)];
				}
			}
			if(NEIGHBOR == 8)
			{
				test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) - 1] = data_index[i] - 1;
				test_index[(NEIGHBOR/2) + te * (NEIGHBOR+1) + 1] = data_index[i] + 1;
				for(int j0=0;j0<3;j0++){
					test_index[j0 + te * (NEIGHBOR+1)] = data_index[i] - 1 - x + j0;
					test_index[j0+6 + te * (NEIGHBOR+1)] = data_index[i] - 1 + x + j0;
				}

				if((data_index[i] % x) == 0){//first row
					for (int j=0; j<3; j++)
						test_index[j*3 + te*(NEIGHBOR+1)] = test_index[j*3+2 + te*(NEIGHBOR+1)];
				}
				if((data_index[i] % x) == (x-1)){//last row
					for(int j=0;j<3;j++)
						test_index[j*3+2 + te*(NEIGHBOR+1)] = test_index[j*3 + te*(NEIGHBOR+1)];
				}
				if((data_index[i]/x) == 0){//first column
					for(int j=0;j<3;j++)
						test_index[j + te*(NEIGHBOR+1)] = test_index[j+6 + te*(NEIGHBOR+1)];
				}
				if((data_index[i]/x) == (y-1)){//last column
					for(int j=0;j<3;j++)
						test_index[j+6  + te*(NEIGHBOR+1)] = test_index[j + te*(NEIGHBOR+1)];
				}
			}

			//int mid = int(labels[data_index[i]])-1 + te*NEU_NUM2;
			test_labels[te] = labels[data_index[i]];
			te = te + 1;
		}
	}
	
	shuffle(train_index, processed_labels, (NEIGHBOR+1), train_size);//shuffle the samples in training set

	//malloc GPU memory, copy data to GPU
	SAFE_CALL(hipMalloc((void **) &gpu_data, sizeof(double) * x * y * z));
	SAFE_CALL(hipMemcpy(gpu_data, data, sizeof(double)* x * y * z, hipMemcpyHostToDevice));

	SAFE_CALL(hipMalloc((void **) &gpu_train_index, sizeof(int) * train_size * (NEIGHBOR+1)));
	SAFE_CALL(hipMemcpy(gpu_train_index, train_index, sizeof(int) * train_size * (NEIGHBOR+1), hipMemcpyHostToDevice));
	SAFE_CALL(hipMalloc((void **) &gpu_test_index, sizeof(int) * test_size * (NEIGHBOR+1)));
	SAFE_CALL(hipMemcpy(gpu_test_index, test_index, sizeof(int) * test_size * (NEIGHBOR+1), hipMemcpyHostToDevice));

	SAFE_CALL(hipMalloc((void **) &gpu_processed_test, sizeof(double) * test_size * (NEIGHBOR+1) * z));
    SAFE_CALL(hipMalloc((void **) &gpu_processed_train, sizeof(double) * train_size * (NEIGHBOR+1) *z));

    delete [] data_index;
    delete [] train_index;
    delete [] test_index;

    int gridsize = 64;
    int blocksize = 512;
	int iter=0;

	processing<<<gridsize,blocksize>>>(iter, gpu_data, gpu_train_index, gpu_processed_train, x, y, z, train_size);
	processing<<<gridsize,blocksize>>>(iter, gpu_data, gpu_test_index, gpu_processed_test, x, y, z, test_size);
	//processing<<<gridsize,blocksize>>>(iter, gpu_data, gpu_valid_index, gpu_processed_valid, x, y, z, valid_size);

	//hipDeviceSynchronize();
	end = clock();
	double tt = double(end - start);
	fprintf(stdout,"Preprocessing Done. (%lfs)\n",tt/CLOCKS_PER_SEC);

	SAFE_CALL(hipFree(gpu_data));
	SAFE_CALL(hipFree(gpu_train_index));
	SAFE_CALL(hipFree(gpu_test_index));
	//hipDeviceSynchronize();
    
	//forward pass
	double * filters = new double [(NEIGHBOR+1)*P_NUM*FILTER_NUM];

	//random initialize 
	for(int i=0; i<(NEIGHBOR+1)*P_NUM*FILTER_NUM; i++){
		filters[i] = 2*(rand()/(double)(RAND_MAX)) - 1 ;
		filters[i] = filters[i]/20;
		if(filters[i] == 0 )
			filters[i] = 0.005;
	}
	
	//count number of convolutional results
	int re_size = 0;
	for (int i=0; i+P_NUM-1<z; i+=LEAP){
		re_size ++;
	}

	
	fprintf(stdout,"re_size:%d\n",re_size);

	int mre_num = (re_size-1)/GP_NUM+1;
	fprintf(stdout,"mre_num:%d\n",mre_num);
	int mre_size = mre_num * FILTER_NUM;
	int ome_num1 = mre_num * FILTER_NUM * NEU_NUM1;//number of weights in full connection layer
	int ome_num2 = NEU_NUM1 * NEU_NUM2;//number of weights in output layer
	
	double * gpu_filters;
	double * gpu_bias0;
	double * gpu_re;//results of concolution layer
	double * gpu_mre;//results of maxpooling layer
	int * gpu_mre_index;//index of max value in each group
	double * gpu_omega1;//weighs of full connection layer
	double * gpu_F1;//outputs of full connection layer
	double * gpu_bias1;
	double * gpu_omega2;
	double * gpu_O2;
	double * gpu_bias2;
	double * gpu_delta_Lz;
	double * gpu_delta_fz;
	double * gpu_delta_fw;
	double * gpu_delta_mw;	
	double * gpu_delta_22;
	double * gpu_delta_kb;
	double * gpu_delta_kw;
    double * gpu_loss_values;

	//copy labels to GPU
	SAFE_CALL(hipMalloc((void**) &gpu_processed_labels, sizeof(double) * train_size * NEU_NUM2));
	SAFE_CALL(hipMemcpy(gpu_processed_labels , processed_labels, sizeof(double) * train_size * NEU_NUM2, hipMemcpyHostToDevice));
	//copy filters to GPU
	SAFE_CALL(hipMalloc((void**) &gpu_filters,sizeof(double) * (NEIGHBOR+1) * P_NUM * FILTER_NUM));
	SAFE_CALL(hipMemcpy(gpu_filters, filters, sizeof(double) * (NEIGHBOR+1) * P_NUM * FILTER_NUM,hipMemcpyHostToDevice));
	//malloc GPU memory for convolutional results
	SAFE_CALL(hipMalloc((void **) &gpu_re,sizeof(double) * re_size * FILTER_NUM * DATA_BATCH));
	//malloc GPU memory for delta_Lz
	SAFE_CALL(hipMalloc((void **) &gpu_delta_Lz, sizeof(double) * NEU_NUM2 * DATA_BATCH));

	//delta_f in full connection layer
	SAFE_CALL(hipMalloc((void **) &gpu_delta_fz, sizeof(double) * NEU_NUM1 * DATA_BATCH));
	SAFE_CALL(hipMalloc((void **) &gpu_delta_fw, sizeof(double) * NEU_NUM1 * NEU_NUM2 * DATA_BATCH));

	//maxpooling
	SAFE_CALL(hipMalloc((void **) &gpu_delta_mw, sizeof(double) * mre_size * NEU_NUM1 * DATA_BATCH));

	//delta in input layer
	SAFE_CALL(hipMalloc((void **) &gpu_delta_22,sizeof(double) * re_size * FILTER_NUM * DATA_BATCH));
	SAFE_CALL(hipMalloc((void **) &gpu_delta_kw, sizeof(double) * (NEIGHBOR+1) * P_NUM * FILTER_NUM * DATA_BATCH));
	SAFE_CALL(hipMalloc((void **) &gpu_delta_kb, sizeof(double) * FILTER_NUM * DATA_BATCH));
    SAFE_CALL(hipMalloc((void **) &gpu_loss_values, sizeof(double) * DATA_BATCH));

    delete [] processed_labels;
	double * omega1 = new double [ome_num1];
	double * omega2 = new double [ome_num2];
	double * bias0 = new double [FILTER_NUM];
	double * bias1 = new double [NEU_NUM1];
	double * bias2 = new double [NEU_NUM2];

	//Initialize omega1
	for(int i=0; i<ome_num1; i++){
		omega1[i] = 2 * (rand()/(double)(RAND_MAX)) - 1;
		omega1[i] = omega1[i]/20;
	        if(omega1[i] == 0)
			omega1[i] = 0.01;
	}
	//initialize bias0
    for(int i=0; i<FILTER_NUM; i++){
		bias0[i] = 2*(rand()/(double)(RAND_MAX)) - 1;
		bias0[i] = bias0[i]/20;
	}
	//initialize bias1
	for(int i=0; i<NEU_NUM1; i++){
		bias1[i] = 2*(rand()/(double)(RAND_MAX)) - 1;
		bias1[i] = bias1[i]/20;
	}

	//initialize omega2
	for(int i=0; i<ome_num2; i++){
		omega2[i] = 2 * (rand()/(double)(RAND_MAX)) - 1;
		omega2[i] = omega2[i]/20;
		if(omega2[i] ==0)
			omega2[i] = 0.01;
	}

	//initialize bias2
	for(int i=0; i<NEU_NUM2; i++){
		bias2[i] = 2*(rand()/(double)(RAND_MAX)) - 1;
		bias2[i] = bias2[i]/20;
	}

    //malloc GPU memory for network parameters and intermediate results, copy the initialized values to GPU
	SAFE_CALL(hipMalloc((void **) &gpu_mre, sizeof(double) * mre_num * FILTER_NUM * DATA_BATCH));
	SAFE_CALL(hipMalloc((void **) &gpu_mre_index, sizeof(int) * mre_num * FILTER_NUM * DATA_BATCH));
	SAFE_CALL(hipMalloc((void **) &gpu_omega1, sizeof(double) * ome_num1));
	SAFE_CALL(hipMalloc((void **) &gpu_omega2, sizeof(double) * ome_num2));
	SAFE_CALL(hipMalloc((void **) &gpu_F1, sizeof(double) * NEU_NUM1 * DATA_BATCH));
	SAFE_CALL(hipMalloc((void **) &gpu_O2, sizeof(double) * NEU_NUM2 * DATA_BATCH));
	SAFE_CALL(hipMalloc((void **) &gpu_bias0, sizeof(double) * FILTER_NUM));
	SAFE_CALL(hipMalloc((void **) &gpu_bias1, sizeof(double) * NEU_NUM1));
	SAFE_CALL(hipMalloc((void **) &gpu_bias2, sizeof(double) * NEU_NUM2));
	SAFE_CALL(hipMemcpy(gpu_omega1, omega1, sizeof(double) * ome_num1, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_omega2, omega2, sizeof(double) * ome_num2, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_bias0, bias0, sizeof(double) * FILTER_NUM, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_bias1, bias1, sizeof(double) * NEU_NUM1, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_bias2, bias2, sizeof(double) * NEU_NUM2, hipMemcpyHostToDevice));



	double * O2 = new double [NEU_NUM2 * DATA_BATCH];//save output value of the network on Host
	double loss;
    double * logloss = new double [1000]();
    double * loss_values = new double [DATA_BATCH];
	double * correct_rate = new double [VALID_BATCH];
    	for(int i=0; i<VALID_BATCH; i++){
        	correct_rate[i] = 1;
    	}

 	double cur_min = 1;
	int count=1;
	int batch_size = 0;
	int batch_num = (train_size - 1)/DATA_BATCH + 1;//count how many bathes are needed to complete the whole training set
	fprintf(stdout,"batch_num:%d\n",batch_num);
	start = clock();
	//creat CUDA streams
	hipStream_t stream[DATA_BATCH];
	for(int i=0; i<DATA_BATCH; i++){
		hipStreamCreate(&stream[i]);
	}
	//batch_num=4;
	//clock_t layer_start,layer_stop;
	//float layer_time ;
	for(int j=0; j<300; j++){
		loss = 0;
		clock_t iter_start = clock();
		for(int i0=0; i0<batch_num; i0++)
		{
			//compute the number of streams(or batch size)
			batch_size = DATA_BATCH;
			if((i0+1 == batch_num) && (train_size%DATA_BATCH != 0))
				batch_size = train_size%DATA_BATCH;
			for(int i1=0; i1<batch_size; i1++)
			{
				// forward propagation
				
				//hipDeviceSynchronize();
				//layer_start=clock();
				convolution<<<FILTER_NUM, re_size, (NEIGHBOR+1) * z * sizeof(double), stream[i1]>>>(i0*DATA_BATCH+i1,i1,gpu_processed_train,gpu_filters,gpu_re,gpu_bias0,z,re_size);
				//hipDeviceSynchronize();
				//layer_stop=clock();
				//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
				//cout<<"conv layer time is "<<layer_time<<" s"<<endl;
				
				//layer_start=clock();
				maxpooling<<<FILTER_NUM,mre_num,0,stream[i1]>>>(i1,gpu_re,gpu_mre,gpu_mre_index,re_size,mre_num);
				//hipDeviceSynchronize();
				//layer_stop=clock();
				//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
				//cout<<"pool layer time is "<<layer_time<<" s"<<endl;
				
				//layer_start=clock();
				fully_connect<<<NEU_NUM1,mre_size, mre_size * sizeof(double), stream[i1]>>>(i1,gpu_mre,gpu_omega1,gpu_bias1,gpu_F1,mre_size);
				//hipDeviceSynchronize();
				//layer_stop=clock();
				//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
				//cout<<"ip1 layer time is "<<layer_time<<" s"<<endl;
				
				//layer_start=clock();
				output<<<1,NEU_NUM2,0,stream[i1]>>>(i1,gpu_F1,gpu_omega2,gpu_bias2,gpu_O2);
				//hipDeviceSynchronize();
				//layer_stop=clock();
				//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
				//cout<<"output layer time is "<<layer_time<<" s"<<endl;
				
				//backward propagation
				//layer_start=clock();
				bp_output<<<1,NEU_NUM2,0,stream[i1]>>>(iter,i0*DATA_BATCH+i1,i1,LEARN_RATE,gpu_processed_labels,gpu_O2,gpu_delta_Lz);
				//hipDeviceSynchronize();
				//layer_stop=clock();
				//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
				//cout<<"bp_output layer time is "<<layer_time<<" s"<<endl;
				
				//layer_start=clock();
				bp_fully_connect<<<NEU_NUM1,NEU_NUM2,0,stream[i1]>>>(iter,i1,LEARN_RATE,gpu_omega2,gpu_F1,gpu_delta_Lz,gpu_delta_fw,gpu_delta_fz);
				//hipDeviceSynchronize();
				//layer_stop=clock();
				//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
				//cout<<"bp_ip layer time is "<<layer_time<<" s"<<endl;
				
				//layer_start=clock();
				bp_maxpooling<<<mre_size,NEU_NUM1,0,stream[i1]>>>(iter,mre_size,re_size,i1,LEARN_RATE,gpu_mre_index,gpu_omega1,gpu_mre,gpu_delta_fz,gpu_delta_mw,gpu_delta_22);
				//hipDeviceSynchronize();
				//layer_stop=clock();
				//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
				//cout<<"bp_pool layer time is "<<layer_time<<" s"<<endl;
				
				//layer_start=clock();
				bp_convolution<<<FILTER_NUM, (NEIGHBOR+1)*P_NUM, (NEIGHBOR+1) * z * sizeof(double), stream[i1]>>>(iter,i0*DATA_BATCH+i1,i1,LEARN_RATE,z,mre_num,re_size,gpu_mre_index,gpu_delta_22,gpu_delta_kw,gpu_delta_kb,gpu_processed_train);
				//hipDeviceSynchronize();
				//layer_stop=clock();
				//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
				//cout<<"bp_conv layer time is "<<layer_time<<" s"<<endl;
			}
			//layer_start=clock();
           loss_function<<<1, batch_size>>>(i0, batch_size, gpu_O2, gpu_processed_labels, gpu_loss_values);
		  //hipDeviceSynchronize();
			//layer_stop=clock();
			//layer_time = float(layer_stop - layer_start) / CLOCKS_PER_SEC;
			//cout<<"loss layer time is "<<layer_time<<" s"<<endl;
            SAFE_CALL(hipMemcpy(loss_values, gpu_loss_values, sizeof(double) * batch_size, hipMemcpyDeviceToHost));
			
			for(int j0=0; j0<batch_size; j0++){
				loss = loss + loss_values[j0];
			}

			//update parameters
			
			update_output<<<NEU_NUM2,batch_size>>>(iter, batch_size, LEARN_RATE, gpu_delta_Lz, gpu_bias2);
			
			update_fully_connect<<<NEU_NUM1,NEU_NUM2>>>(batch_size, LEARN_RATE, gpu_omega2, gpu_bias1, gpu_delta_fw, gpu_delta_fz);
			
			update_maxpooling<<<mre_size,NEU_NUM1>>>(iter, mre_size, batch_size, LEARN_RATE, gpu_omega1, gpu_delta_mw);
			
			update_convolution<<<FILTER_NUM,(NEIGHBOR+1)*P_NUM>>>(batch_size, re_size, LEARN_RATE, gpu_delta_22, gpu_delta_kw, gpu_delta_kb, gpu_filters, gpu_bias0);
			
        }
		 clock_t iter_stop = clock();
        		float iter_time = float(iter_stop - iter_start) / CLOCKS_PER_SEC;
		
		double single_rate = loss/train_size;
       		logloss[j] = single_rate;
		char str[50];
        	sprintf(str, "%d", j + 1);
        	strcat(str, ",");
		fprintf(stdout,"[Cube CNN training with MBGD algo.  BatchSize = %d  Execution time: %.3f sec] Iteration %-4s loss = %lf;\n", 
                DATA_BATCH, iter_time, str, single_rate);
		//fprintf(stdout,"Iteration %d,	loss = %lf;\n",j+1,single_rate);
        	
		insert_line(correct_rate,single_rate);//insert current loss into the line
		double new_min = *min_element(correct_rate, correct_rate + VALID_BATCH);
        	if(cur_min > new_min){
            		cur_min = new_min;
		     	count = 1;
        	}
        	else{
            		count++;
        	}
        	if(count >= VALID_BATCH ) {
            		LEARN_RATE = LEARN_RATE * 0.9;
            		fprintf(stdout,"LEARN_RATE:%lf\n",LEARN_RATE);
            		count = 1;
            		cur_min = new_min;
        	}
        	if(single_rate < MIN_ERR)
            		break;
	}

	fprintf(stdout,"Training completed!\n");
	end = clock();
	tt = double(end - start);
	fprintf(stdout,"Exesution time of training:%lfs\n",tt/CLOCKS_PER_SEC);

	start = clock();
	hipDeviceSynchronize();
	SAFE_CALL(hipMemcpy(filters, gpu_filters, sizeof(double) * (NEIGHBOR+1) * P_NUM * FILTER_NUM, hipMemcpyDeviceToHost));
	SAFE_CALL(hipMemcpy(bias0, gpu_bias0, sizeof(double) * FILTER_NUM, hipMemcpyDeviceToHost));
	SAFE_CALL(hipMemcpy(bias1, gpu_bias1, sizeof(double) * NEU_NUM1, hipMemcpyDeviceToHost));
	SAFE_CALL(hipMemcpy(bias2, gpu_bias2, sizeof(double) * NEU_NUM2, hipMemcpyDeviceToHost));
	SAFE_CALL(hipMemcpy(omega1, gpu_omega1, sizeof(double) * ome_num1, hipMemcpyDeviceToHost));
	SAFE_CALL(hipMemcpy(omega2, gpu_omega2, sizeof(double) * ome_num2, hipMemcpyDeviceToHost));
	
	//write the network parameters into a mat file
	MATFile * pmatFile;
	pmatFile = matOpen("model.mat","w");
	mxArray * m1 = mxCreateDoubleMatrix((NEIGHBOR+1)*P_NUM, FILTER_NUM,mxREAL);
	memcpy((void *)mxGetPr(m1), (void *)filters, sizeof(double) * (NEIGHBOR+1) * P_NUM * FILTER_NUM);
	matPutVariable(pmatFile, "filters", m1);

	mxArray * m2 = mxCreateDoubleMatrix(FILTER_NUM,1,mxREAL);
	memcpy((void *)mxGetPr(m2), (void *)bias0, sizeof(double) * FILTER_NUM);
	matPutVariable(pmatFile, "bias0", m2);

	mxArray * m3 = mxCreateDoubleMatrix(NEU_NUM1,mre_size,mxREAL);
	memcpy((void *)mxGetPr(m3), (void *)omega1, sizeof(double) * ome_num1);
	matPutVariable(pmatFile, "omega1", m3);

	mxArray * m4 = mxCreateDoubleMatrix(NEU_NUM1,1,mxREAL);
       	memcpy((void *)mxGetPr(m4), (void *)bias1, sizeof(double) * NEU_NUM1);
	matPutVariable(pmatFile, "bias1", m4);

	mxArray * m5 = mxCreateDoubleMatrix(NEU_NUM2,NEU_NUM1,mxREAL);
	memcpy((void *)mxGetPr(m5), (void *)omega2, sizeof(double) * ome_num2);
	matPutVariable(pmatFile, "omega2", m5);

	mxArray * m6 = mxCreateDoubleMatrix(NEU_NUM2,1,mxREAL);
	memcpy((void *)mxGetPr(m6), (void *)bias2, sizeof(double) * NEU_NUM2);
	matPutVariable(pmatFile, "bias2", m6);

    mxArray * m7 = mxCreateDoubleMatrix(300,1,mxREAL);
    memcpy((void *)mxGetPr(m7), (void *)logloss, sizeof(double) * 300);
    matPutVariable(pmatFile, "loss", m7);

	matClose(pmatFile);

    delete [] filters;
    delete [] bias0;
    delete [] omega1;
    delete [] bias1;
    delete [] omega2;
    delete [] bias2;
    delete [] logloss;
    delete [] loss_values;
    delete [] correct_rate;

	for(int i=0; i<DATA_BATCH; i++){
		hipStreamDestroy(stream[i]);
	}
	
	//test
	double right = 0;
	double count0 = 0;
    hipStream_t testStream[test_size]; 
    for (size_t i=0; i<test_size; i++){
        hipStreamCreate(&testStream[i]);
    }

	for (int i1=0; i1<test_size; i1++){
		convolution<<<FILTER_NUM, re_size, (NEIGHBOR+1) * z * sizeof(double), testStream[i1]>>>(i1,0,gpu_processed_test,gpu_filters,gpu_re,gpu_bias0,z,re_size);
		//hipDeviceSynchronize();

		maxpooling<<<FILTER_NUM, mre_num, 0, testStream[i1]>>>(0,gpu_re,gpu_mre,gpu_mre_index,re_size,mre_num);
		//hipDeviceSynchronize();

		fully_connect<<<NEU_NUM1,mre_size, mre_size * sizeof(double), testStream[i1]>>>(0,gpu_mre,gpu_omega1,gpu_bias1,gpu_F1,mre_size);
		//hipDeviceSynchronize();

		output<<<1,NEU_NUM2, 0, testStream[i1]>>>(0,gpu_F1,gpu_omega2,gpu_bias2,gpu_O2);
		//hipDeviceSynchronize();

		SAFE_CALL(hipMemcpy(O2, gpu_O2, sizeof(double) * NEU_NUM2, hipMemcpyDeviceToHost));
		//hipDeviceSynchronize();

		right = count_err(test_labels, O2, i1);
		count0 = count0 + right;
	}

    delete [] O2;
    delete [] test_labels;

    hipFree(gpu_processed_test);
    hipFree(gpu_mre);
    hipFree(gpu_filters);
    hipFree(gpu_re);
    hipFree(gpu_bias0);
    hipFree(gpu_mre_index);
    hipFree(gpu_omega1);
    hipFree(gpu_bias1);
    hipFree(gpu_F1);
    hipFree(gpu_omega2);
    hipFree(gpu_bias2);
    hipFree(gpu_O2);

    for (size_t i=0; i<test_size; i++){
        hipStreamDestroy(testStream[i]);
    }
	end = clock();
	tt = double(end - start);
	fprintf(stdout,"Execution time of testing:%lfs\n",tt/CLOCKS_PER_SEC);
	return count0/test_size;
}


int main(int argc, char * argv[])
{
  	if(!InitCUDA()){
		return 0;
	}
	printf("CUDA initialized.\n");

	clock_t start,end;

	double *trainset,*trainlabels;
	if(argc!=3){
		fprintf(stderr, "3 input arguments required!");
		return 0;
	}

	MATFile * datamat = matOpen(argv[1], "r");
mxArray * train = matGetVariable(datamat,"KSC");
	MATFile * labelmat = matOpen(argv[2], "r");
mxArray * labels = matGetVariable(labelmat,"KSC_gt");

	trainset = (double*)mxGetData(train);
	trainlabels = (double*)mxGetData(labels);

	const mwSize * dim;
	dim = mxGetDimensions(train);
	matClose(datamat);

	start = clock();
	double correct = training(trainset, trainlabels, dim[0], dim[1], dim[2]);
	end = clock();
	fprintf(stdout,"Correct Rate:%lf(300 iterations, train:test=4:1)\n",correct);
	double usetime = double(end - start);
	fprintf(stdout, "Execution time of the whole program:%lfs\n",usetime/CLOCKS_PER_SEC);
    hipDeviceReset();
	return 0;
}
